#include "railgun.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <queue>
#include <gc.h>

std::queue<railgun_task*> tq;
#define malloc GC_malloc
#define realloc GC_realloc
#define calloc(m,n) GC_malloc((m)*(n))
#define free

int
_schedule(void* f, railgun_args* args, dim3 blocks, dim3 threads)
{
  railgun_task* t;

  t = (railgun_task*)malloc(sizeof(railgun_task));

  t->f = f;
  t->args = args;
  t->blocks = blocks;
  t->threads = threads;

  tq.push(t);

  return 0;
}

size_t
get_data_size(railgun_data_type t)
{
  switch (t) {
  case RG_TYPE_INT_P:
  case RG_TYPE_INT:
    return sizeof(int);
  case RG_TYPE_FLOAT_P:
  case RG_TYPE_FLOAT:
    return sizeof(float);
  case RG_TYPE_DOUBLE_P:
  case RG_TYPE_DOUBLE:
    return sizeof(double);
  default:
    return 0;
  }
}

// typedef void (*iii_f)(int*, int*, int*);
// typedef void (*ii_f)(int*, int*);
// typedef void (*dd_f)(double*, double*);
// typedef void (*ddd_f)(double*, double*, double*);
// typedef void (*Iddd_f)(int, double*, double*, double*);

void
execute_task(railgun_task* t, railgun_memory* mem, hipStream_t* strm)
// execute_task(railgun_task* t, railgun_memory* mem)
{
  hipError_t err = hipSuccess;
  int i, argc;
  size_t size;
  railgun_args *args;
  railgun_data *argv, *d;

  args = t->args;
  argc = args->argc;
  argv = args->argv;

  for (i = 0; i < argc; i++) {
    d = &argv[i];
    size = d->n * get_data_size(d->type);
    switch (d->type) {
      case RG_TYPE_FLOAT_P:
        hipMalloc((void**)&(mem[i].fp), size);
        if (d->dir == RG_DIR_DOWNLOAD)
          hipMemcpyAsync(mem[i].fp, d->d.fp, size, hipMemcpyHostToDevice, *strm);
          // hipMemcpy(mem[i].fp, d->d.fp, size, hipMemcpyHostToDevice);
        break;
      case RG_TYPE_DOUBLE_P:
        hipMalloc((void**)&(mem[i].dp), size);
        if (d->dir == RG_DIR_DOWNLOAD)
          hipMemcpyAsync(mem[i].dp, d->d.dp, size, hipMemcpyHostToDevice, *strm);
          // hipMemcpy(mem[i].dp, d->d.dp, size, hipMemcpyHostToDevice);
        break;
      case RG_TYPE_INT:
        mem[i].i = d->d.i;
        break;
      default:
        break;
    }
  }

  // err = hipMalloc((void**)&da, argv[1].n * sizeof(float));
  // err = hipMalloc((void**)&db, argv[2].n * sizeof(float));
  // err = hipMalloc((void**)&dc, argv[3].n * sizeof(float));
  //
  // err = hipMemcpy(da, argv[1].d.fp, argv[1].n * sizeof(float), hipMemcpyHostToDevice);
  // err = hipMemcpy(db, argv[2].d.fp, argv[2].n * sizeof(float), hipMemcpyHostToDevice);

  // ((void (*)(int,float*,float*,float*))t.f)<<<t.blocks, t.threads>>>(mem[0].i, mem[1].fp, mem[2].fp, mem[3].fp);
  printf("now, the execution will start\n");
  // _execute_kernel(args->fmt, t, mem, strm);
  // _execute_kernel(args->fmt, t, mem);
  ((void (*)(int,float*,float*,float*))t->f)<<<t->blocks, t->threads, 0, *strm>>>(mem[0].i, mem[1].fp, mem[2].fp, mem[3].fp);
  // ((void (*)(int,float*,float*,float*))t->f)<<<t->blocks, t->threads>>>(mem[0].i, mem[1].fp, mem[2].fp, mem[3].fp);
  // ((void (*)(int,double*,double*,double*))t->f)<<<t->blocks, t->threads>>>(mem[0].i, mem[1].dp, mem[2].dp, mem[3].dp);

  for (i = 0; i < argc; i++) {
    d = &argv[i];
    if (d->dir == RG_DIR_READBACK) {
      size = d->n * get_data_size(d->type);
      switch (d->type) {
        case RG_TYPE_FLOAT_P:
          hipMemcpyAsync(d->d.fp, mem[i].fp, size, hipMemcpyDeviceToHost, *strm);
          // hipMemcpy(d->d.fp, mem[i].fp, size, hipMemcpyDeviceToHost);
          break;
        case RG_TYPE_DOUBLE_P:
          hipMemcpyAsync(d->d.dp, mem[i].dp, size, hipMemcpyDeviceToHost, *strm);
          // hipMemcpy(d->d.dp, mem[i].dp, size, hipMemcpyDeviceToHost);
          break;
        default:
          break;
      }
    }
  }
  // err = hipMemcpy(argv[3].d.fp, dc, argv[3].n * sizeof(float), hipMemcpyDeviceToHost);


  return;
}

void
wait_streams(hipStream_t* strms, int n)
{
  int i;

  for (i = 0; i < n; i++) {
    hipStreamSynchronize(strms[i]);
  }

  return;
}

int
_execute()
{
  int i, j, n, total, max_i;
  int totals[10];
  railgun_data d;
  railgun_task *t;
  railgun_task *tasks[10];
  railgun_memory **mems;
  hipStream_t *strms;

  j = 0;
  while (!tq.empty()) {
    t = tq.front();
    tq.pop();
    tasks[j] = t;

    total = 0;
    for (i = 0; i < t->args->argc; i++) {
      d = t->args->argv[i];
      total += get_data_size(d.type) * d.n;
    }
    totals[j] = total;
    j++;
  }
  n = j;
  mems = (railgun_memory**)malloc(n * sizeof(railgun_memory*));
  strms = (hipStream_t*)malloc(n * sizeof(hipStream_t));
  for (j = 0; j < n; j++) {
    max_i = 0;
    for (i = 0; i < n; i++) {
      if (totals[i] > totals[max_i]) {
        max_i = i;
      }
    }

    // printf("%d\n", totals[max_i]);
    // printf("%d\n", max_i);

    hipStreamCreate(&strms[max_i]);
    t = tasks[max_i];
    mems[max_i] = (railgun_memory*)malloc(t->args->argc * sizeof(railgun_memory*));
    execute_task(t, mems[max_i], &strms[max_i]);

    totals[max_i] = 0;
  }

  wait_streams(strms, n);

  // free railgun_memory(on GPU) and stream
  for (i = 0; i < n; i++) {
    // free(mems[i]);
    hipStreamDestroy(strms[i]);
  }
  // free(mems);
  free(strms);

  return 0;
}
