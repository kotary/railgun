#include "railgun.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <queue>
#include <gc.h>

#define malloc GC_malloc
#define realloc GC_realloc
#define calloc(m,n) GC_malloc((m)*(n))
#define free(x)

std::queue<railgun_task> tq;

size_t
get_data_size(railgun_data_type t)
{
  switch (t) {
  case RG_TYPE_INT_P:
  case RG_TYPE_INT:
    return sizeof(int);
  case RG_TYPE_FLOAT_P:
  case RG_TYPE_FLOAT:
    return sizeof(float);
  case RG_TYPE_DOUBLE_P:
  case RG_TYPE_DOUBLE:
    return sizeof(double);
  default:
    return 0;
  }
}

int
_schedule(void* f, railgun_args* args, dim3 blocks, dim3 threads)
{
  railgun_task *t;
  railgun_data *d;
  int i;

  t = (railgun_task*)malloc(sizeof(railgun_task));
  t->f = f;
  t->args = args;
  t->blocks = blocks;
  t->threads = threads;
  t->total = 0;

  d = t->args->argv;
  for (i = 0; i < t->args->argc; i++) {
    t->total += d[i].n * get_data_size(d[i].type);
  }

  bheap_push(task_q, t->total, t);

  return 0;
}

void
kernel_call(railgun_task* t, railgun_memory* mem, hipStream_t* strm)
{
  // printf("Kernel Execution: %s\n", t->args->fmt);
  if (!strcmp(t->args->fmt, "Iff")) {
    ((void (*)(int,float*,float*))t->f)<<<t->blocks, t->threads, 0, *strm>>>(mem[0].i, mem[1].fp, mem[2].fp);
  } else if (!strcmp(t->args->fmt, "Ifff")) {
    ((void (*)(int,float*,float*,float*))t->f)<<<t->blocks, t->threads, 0, *strm>>>(mem[0].i, mem[1].fp, mem[2].fp, mem[3].fp);
  }
  return;
}

void
data_upload(railgun_data* d, railgun_memory* mem, hipStream_t* strm)
{
  int size;

  size = d->n * get_data_size(d->type);
  switch (d->type) {
    case RG_TYPE_FLOAT_P:
      hipMalloc((void**)&(mem->fp), size);
      if (d->dir == RG_DIR_DOWNLOAD)
        hipMemcpyAsync(mem->fp, d->d.fp, size, hipMemcpyHostToDevice, *strm);
      break;
    case RG_TYPE_DOUBLE_P:
      hipMalloc((void**)&(mem->dp), size);
      if (d->dir == RG_DIR_DOWNLOAD)
        hipMemcpyAsync(mem->dp, d->d.dp, size, hipMemcpyHostToDevice, *strm);
      break;
    case RG_TYPE_INT:
      mem->i = d->d.i;
      break;
    default:
      break;
  }
  return;
}

void
data_download(railgun_data* d, railgun_memory* mem, hipStream_t* strm)
{
  int size;
  if (d->dir == RG_DIR_READBACK) {
    size = d->n * get_data_size(d->type);
    switch (d->type) {
      case RG_TYPE_FLOAT_P:
        hipMemcpyAsync(d->d.fp, mem->fp, size, hipMemcpyDeviceToHost, *strm);
        break;
      case RG_TYPE_DOUBLE_P:
        hipMemcpyAsync(d->d.dp, mem->dp, size, hipMemcpyDeviceToHost, *strm);
        break;
      default:
        break;
    }
  }
  return;
}

void execute_tasks_df(int n, railgun_task* ts, railgun_memory** mems, hipStream_t *strms)
{
  int i, j, size;
  railgun_data *d, *argv;

  for (i = 0; i < n; i++) {
    // Phase 00: Pre-Processing

    // Phase 01: Data Transfer
    for (j = 0; j < ts[i].args->argc; j++) {
      data_upload(&ts[i].args->argv[j], &mems[i][j], &strms[i]);
    }

    // Phase 02: Kernel Execution
    // ((void (*)(int,float*,float*,float*))ts[i].f)<<<ts[i].blocks, ts[i].threads, 0, strms[i]>>>(mems[i][0].i, mems[i][1].fp, mems[i][2].fp, mems[i][3].fp);
    kernel_call(&ts[i], mems[i], &strms[i]);

    // Phase 03: Data Transfer(GPU -> CPU)
    for (j = 0; j < ts[i].args->argc; j++) {
      data_download(&ts[i].args->argv[j], &mems[i][j], &strms[i]);
    }

  }
}

void
execute_tasks_bf(int n, railgun_task* ts, railgun_memory** mems, hipStream_t* strms)
{
  int i, j, argc;
  hipError_t err = hipSuccess;
  size_t size;
  railgun_args *args;
  railgun_data *argv, *d;

  // Phase 00: Pre-Processing

  // Phase 01: Data Transfer(CPU -> GPU)
  for (i = 0; i < n; i++) {
    for (j = 0; j < ts[i].args->argc; j++) {
      data_upload(&ts[i].args->argv[j], &mems[i][j], &strms[i]);
    }
  }


  // Phase 02: Kernel Execution
  for (i = 0; i < n; i++) {
    kernel_call(&ts[i], mems[i], &strms[i]);
  }

  // Phase 03: Data Transfer(GPU -> CPU)
  for (i = 0; i < n; i++) {
    for (j = 0; j < ts[i].args->argc; j++) {
      data_download(&ts[i].args->argv[j], &mems[i][j], &strms[i]);
    }
  }

  // Phase 04: Post-Processing

  return;
}

void
wait_streams(hipStream_t* strms, int n)
{
  int i;

  for (i = 0; i < n; i++) {
    printf("waiting...:%p\n", &(strms[i]));
    hipStreamSynchronize(strms[i]);
  }

  return;
}

void
task_free(railgun_task* t)
{
  free(t->args->fmt);
  free(t->args->argv);
  free(t);
  return;
}

int
_execute()
{
  railgun_task *tasks;
  railgun_data *d;
  railgun_memory **mems;
  hipStream_t *strms;
  int i, j, task_n, total;

  task_n = task_q->tail + 1;
  tasks = (railgun_task*)malloc(task_n * sizeof(railgun_task));
  for (i = 0; i < task_n; i++) {
    tasks[i] = *((railgun_task*)bheap_pop(task_q).opt);
  }

  mems = (railgun_memory**)malloc(task_n * sizeof(railgun_memory*));
  strms = (hipStream_t*)malloc(task_n * sizeof(hipStream_t));

  for (i = 0; i < task_n; i++) {
    mems[i] = (railgun_memory*)malloc(tasks[i].args->argc * sizeof(railgun_memory));
    hipStreamCreate(&(strms[i]));
  }

  // execute_tasks_df(task_n, tasks, mems, strms);
  execute_tasks_bf(task_n, tasks, mems, strms);

  wait_streams(strms, task_n);
  for (i = 0; i < task_n; i++) {
    free(mems[i]);
    hipStreamDestroy(strms[i]);
    task_free(&tasks[i]);
  }
  free(tasks);
  free(mems);
  free(strms);

  return 0;
}
